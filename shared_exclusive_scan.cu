
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define N 512

__global__ void exclusive_scan(int *d_in)
{
    __shared__ int temp_in[N];

    int tid = threadIdx.x;
    temp_in[tid] = d_in[tid];
    
    __syncthreads();

    //Phase 1 (Uptree)
    int s = 1;
    for(; s<=N-1; s<<=1)
    {
        int i = 2*s*(threadIdx.x+1)-1;
        if((i >= s) && (i<N)) {
            //printf("s = %d, i= %d \n", s, i);
            int a = temp_in[i];
            int b = temp_in[i-s];
            __syncthreads();
            temp_in[i] = a+b;
        }
        __syncthreads();

        
    }

    

    //Phase 2 (Downtree)
    if(threadIdx.x == 0)
        temp_in[N-1] = 0;
    
    for(s = s/2; s >= 1; s>>=1)
    {
        int i = 2*s*(threadIdx.x+1)-1;
        if((i >= s) && (i<N)) {
            //printf("s = %d, i= %d \n", s, i);
            int r = temp_in[i];
            int l = temp_in[i-s];
            __syncthreads();
            temp_in[i] = l+r;
            temp_in[i-s] = r;
        }
        __syncthreads();

        
    }

    d_in[tid] = temp_in[tid];
    
}

int main()
{
	int h_in[N];
	int h_out[N];

    for(int i=0; i < N; i++)
        h_in[i] = 1;


	// h_in[0] = 3;
    // h_in[1] = 1;
    // h_in[2] = 7;
    // h_in[3] = 0;
    // h_in[4] = 4;
    // h_in[5] = 1;
    // h_in[6] = 6;
    // h_in[7] = 3;

	int *d_in;
	//int *d_out;

	hipMalloc((void**) &d_in, N*sizeof(int));
	
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);
	
    hipEventRecord(start);

	//Implementing kernel call
	exclusive_scan<<<1, N>>>(d_in);

    hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(&h_out, d_in, N*sizeof(int), hipMemcpyDeviceToHost);

	for(int i=0; i<N; i++)
		printf("out[%d] =  %d\n", i, h_out[i]);

    hipFree(d_in);

    printf("Time used: %f milliseconds\n", milliseconds);
	return -1;

}
